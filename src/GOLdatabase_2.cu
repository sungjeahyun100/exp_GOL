#include "hip/hip_runtime.h"
/**
 * Game of Life Database Generator - d_matrix_2 version
 * Conway's Game of Life 패턴 생성 및 라벨링을 위한 데이터베이스 생성기
 */

#include "GOLdatabase_2.hpp"

namespace GOL_2 {
    using namespace d_matrix_ver2;

    #define MAXGEN 2500
    namespace fs = std::filesystem;

    const int BOARDWIDTH = 100;
    const int BOARDHEIGHT = 100;
    const int WIDTH = 10;
    const int HEIGHT = 10;

    // CUDA kernel: Game of Life 다음 세대 계산
    __global__ void nextGenKernel(int* current, int* next, int width, int height) {
        int i = blockIdx.y * blockDim.y + threadIdx.y;
        int j = blockIdx.x * blockDim.x + threadIdx.x;

        if (i < height && j < width) {
            int alive = 0;
            for (int dx = -1; dx <= 1; ++dx) {
                for (int dy = -1; dy <= 1; ++dy) {
                    if(dx == 0 && dy == 0) continue;
                    int ni = i + dx;
                    int nj = j + dy;
                    if (ni >= 0 && ni < height && nj >= 0 && nj < width) {
                        alive += current[ni * width + nj];
                    }
                }
            }

            int idx = i * width + j;
            if (current[idx] == 1) {
                next[idx] = (alive == 2 || alive == 3) ? 1 : 0;
            } else {
                next[idx] = (alive == 3) ? 1 : 0;
            }
        }
    }

    d_matrix_2<int> nextGen(const d_matrix_2<int>& current, hipStream_t str) {
        d_matrix_2<int> next(current.getRow(), current.getCol(), str);
        int* d_curr = current.getDevPointer();
        int* d_next = next.getDevPointer();

        dim3 blockSize(32, 32);
        dim3 gridSize((current.getCol() + 31) / 32, (current.getRow() + 31) / 32);

        nextGenKernel<<<gridSize, blockSize, 0, str>>>(d_curr, d_next, current.getCol(), current.getRow());
        hipStreamSynchronize(str);
        
        return next;
    }

    __global__ void placePatternKernel(int* board, int* pattern, int fullHeight, int fullWidth,
        int patternHeight, int patternWidth,
        int startRow, int startCol) {
        int i = blockIdx.y * blockDim.y + threadIdx.y; // pattern row
        int j = blockIdx.x * blockDim.x + threadIdx.x; // pattern col

        if (i < patternHeight && j < patternWidth) {
            int boardIdx = (startRow + i) * fullWidth + (startCol + j);
            int patternIdx = i * patternWidth + j;
            board[boardIdx] = pattern[patternIdx];
        }
    }

    d_matrix_2<int> generateFixedRatioPatternWithPadding(int fullHeight, int fullWidth, int patternHeight, int patternWidth, double aliveRatio, hipStream_t str) {
        // 1. CPU에서 pattern 배열 셔플
        int totalPatternCells = patternHeight * patternWidth;
        int aliveCells = static_cast<int>(totalPatternCells * aliveRatio);
        std::vector<int> host_pattern(totalPatternCells, 0);
        std::fill_n(host_pattern.begin(), aliveCells, 1);

        std::random_device rd;
        std::mt19937_64 gen(rd());
        std::shuffle(host_pattern.begin(), host_pattern.end(), gen);

        // 2. GPU 메모리로 복사
        thrust::device_vector<int> d_pattern = host_pattern;
        d_matrix_2<int> board(fullHeight, fullWidth, str); // 전체 보드
        board.fill(0, str); // 0으로 초기화

        int startRow = (fullHeight - patternHeight) / 2;
        int startCol = (fullWidth - patternWidth) / 2;

        // 3. 커널로 중앙에 패턴 복사
        dim3 blockSize(16, 16);
        dim3 gridSize((patternWidth + 15) / 16, (patternHeight + 15) / 16);

        placePatternKernel<<<gridSize, blockSize, 0, str>>>(
            board.getDevPointer(), 
            thrust::raw_pointer_cast(d_pattern.data()), 
            fullHeight, fullWidth, 
            patternHeight, patternWidth, 
            startRow, startCol
        );

        hipStreamSynchronize(str);
        return board;
    }

    d_matrix_2<int> generateFixedRatioPatternWithSeed(int fullHeight, int fullWidth, int patternHeight, int patternWidth, double aliveRatio, int seed, hipStream_t str) {
        // 1. CPU에서 pattern 배열 셔플
        int totalPatternCells = patternHeight * patternWidth;
        int aliveCells = static_cast<int>(totalPatternCells * aliveRatio);
        std::vector<int> host_pattern(totalPatternCells, 0);
        std::fill_n(host_pattern.begin(), aliveCells, 1);

        std::mt19937_64 gen(seed);
        std::shuffle(host_pattern.begin(), host_pattern.end(), gen);

        // 2. GPU 메모리로 복사
        thrust::device_vector<int> d_pattern = host_pattern;
        d_matrix_2<int> board(fullHeight, fullWidth, str); // 전체 보드
        board.fill(0, str); // 0으로 초기화

        int startRow = (fullHeight - patternHeight) / 2;
        int startCol = (fullWidth - patternWidth) / 2;

        // 3. 커널로 중앙에 패턴 복사
        dim3 blockSize(16, 16);
        dim3 gridSize((patternWidth + 15) / 16, (patternHeight + 15) / 16);

        placePatternKernel<<<gridSize, blockSize, 0, str>>>(
            board.getDevPointer(), 
            thrust::raw_pointer_cast(d_pattern.data()), 
            fullHeight, fullWidth, 
            patternHeight, patternWidth, 
            startRow, startCol
        );

        hipStreamSynchronize(str);
        return board;
    }

    __global__ void countAliveKernel(int* mat, int* partialSums, int totalSize) {
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;
        int localSum = 0;

        for (int i = tid; i < totalSize; i += stride) {
            localSum += mat[i];
        }

        if (tid < totalSize) {
            partialSums[tid] = localSum;
        }
    }

    // Fast alive-cell counter using thrust reduction (no per-iteration allocations)
    int countAlive(const d_matrix_2<int>& mat, hipStream_t str) {
        int totalSize = mat.getRow() * mat.getCol();
        thrust::device_ptr<const int> ptr(mat.getDevPointer());
        // Sum 0/1 values directly on device
        int total = thrust::reduce(
            thrust::cuda::par.on(str),
            ptr, ptr + totalSize,
            0, thrust::plus<int>()
        );
        // Ensure reduction is complete before returning
        hipStreamSynchronize(str);
        return total;
    }

    // Optimized simulation using ping-pong device buffers (avoids per-step allocations)
    int simulateAndLabel(const d_matrix_2<int>& initialPattern, int fileId, hipStream_t str) {
        const int H = initialPattern.getRow();
        const int W = initialPattern.getCol();

        // Allocate a second device buffer for next state (track separately)
        int* d_curr = initialPattern.getDevPointer();
        int* d_next = nullptr;
        int* d_alt  = nullptr; // always points to allocated buffer to free later
        hipMallocAsync(&d_alt, sizeof(int) * H * W, str);
        d_next = d_alt;

        std::deque<int> history; // 최근 50개 alive 수 저장
        const int window = 50;

        int constantCount = 0;
        int prev = -1;
        bool strictlyIncreasing = true;
        int gen = 0;

        dim3 blockSize(32, 32);
        dim3 gridSize((W + 31) / 32, (H + 31) / 32);

        while (gen < MAXGEN) {
            // Count alive cells on device
            int alive = 0;
            {
                thrust::device_ptr<const int> ptr(d_curr);
                alive = thrust::reduce(thrust::cuda::par.on(str), ptr, ptr + H * W, 0, thrust::plus<int>());
                hipStreamSynchronize(str);
            }

            // history 갱신
            if (static_cast<int>(history.size()) >= window) history.pop_front();
            history.push_back(alive);

            if (prev == alive) constantCount++;
            else constantCount = 0;

            if (prev != -1 && alive <= prev) strictlyIncreasing = false;
            if (constantCount >= 100 || (strictlyIncreasing && gen >= 100)) break;

            prev = alive;

            // Next generation in-place to d_next, then swap pointers
            nextGenKernel<<<gridSize, blockSize, 0, str>>>(d_curr, d_next, W, H);
            hipStreamSynchronize(str);
            std::swap(d_curr, d_next);
            gen++;
        }

        // Final alive count on the current buffer
        int final_alive = 0;
        {
            thrust::device_ptr<const int> ptr(d_curr);
            final_alive = thrust::reduce(thrust::cuda::par.on(str), ptr, ptr + H * W, 0, thrust::plus<int>());
            hipStreamSynchronize(str);
        }

    if (d_alt) hipFreeAsync(d_alt, str);
        hipStreamSynchronize(str);
        return final_alive;
    }

    d_matrix_2<int> simulateAndLabelingtopattern(const d_matrix_2<int>& initialPattern, int fileId, hipStream_t str) {
        const int H = initialPattern.getRow();
        const int W = initialPattern.getCol();

        int* d_curr = initialPattern.getDevPointer();
        int* d_next = nullptr;
        int* d_alt  = nullptr;
        hipMallocAsync(&d_alt, sizeof(int) * H * W, str);
        d_next = d_alt;

        std::deque<int> history; // 최근 50개 alive 수 저장
        const int window = 50;

        int constantCount = 0;
        int prev = -1;
        bool strictlyIncreasing = true;
        int gen = 0;

        dim3 blockSize(32, 32);
        dim3 gridSize((W + 31) / 32, (H + 31) / 32);

        while (gen < MAXGEN) {
            int alive = 0;
            {
                thrust::device_ptr<const int> ptr(d_curr);
                alive = thrust::reduce(thrust::cuda::par.on(str), ptr, ptr + H * W, 0, thrust::plus<int>());
                hipStreamSynchronize(str);
            }

            if (static_cast<int>(history.size()) >= window) history.pop_front();
            history.push_back(alive);

            if (prev == alive) constantCount++;
            else constantCount = 0;

            if (prev != -1 && alive <= prev) strictlyIncreasing = false;
            if (constantCount >= 100 || (strictlyIncreasing && gen >= 100)) break;

            prev = alive;

            nextGenKernel<<<gridSize, blockSize, 0, str>>>(d_curr, d_next, W, H);
            hipStreamSynchronize(str);
            std::swap(d_curr, d_next);
            gen++;
        }

        // Copy final board into d_matrix_2
        d_matrix_2<int> final_board(H, W, str);
        hipMemcpyAsync(final_board.getDevPointer(), d_curr, sizeof(int) * H * W, hipMemcpyDeviceToDevice, str);
        hipStreamSynchronize(str);

        if (d_alt) hipFreeAsync(d_alt, str);
        hipStreamSynchronize(str);
        return final_board;
    }

    // 최적화: 단일 시뮬레이션으로 패턴과 라벨을 동시에 반환
    std::pair<d_matrix_2<int>, int> simulateAndGetBoth(const d_matrix_2<int>& initialPattern, int fileId, hipStream_t str) {
        const int H = initialPattern.getRow();
        const int W = initialPattern.getCol();

        int* d_curr = initialPattern.getDevPointer();
        int* d_next = nullptr;
        int* d_alt  = nullptr;
        hipMallocAsync(&d_alt, sizeof(int) * H * W, str);
        d_next = d_alt;

        std::deque<int> history; // 최근 50개 alive 수 저장
        const int window = 50;

        int constantCount = 0;
        int prev = -1;
        bool strictlyIncreasing = true;
        int gen = 0;

        dim3 blockSize(32, 32);
        dim3 gridSize((W + 31) / 32, (H + 31) / 32);

        while (gen < MAXGEN) {
            int alive = 0;
            {
                thrust::device_ptr<const int> ptr(d_curr);
                alive = thrust::reduce(thrust::cuda::par.on(str), ptr, ptr + H * W, 0, thrust::plus<int>());
                hipStreamSynchronize(str);
            }

            if (static_cast<int>(history.size()) >= window) history.pop_front();
            history.push_back(alive);

            if (prev == alive) constantCount++;
            else constantCount = 0;

            if (prev != -1 && alive <= prev) strictlyIncreasing = false;

            // 더 빠른 조기 종료: 안정화 감지 개선
            if (constantCount >= 30) break;  // 30 세대 연속 동일 → 안정화
            if (strictlyIncreasing && gen >= 50) break;  // 50 세대 연속 증가 → 발산
            if (alive == 0) break;  // 모든 셀 사망 → 소멸

            // 진동 패턴 감지: 최근 기록에서 반복 확인
            if (history.size() >= 20) {
                bool oscillating = true;
                int period = 2;  // 2주기 진동 확인
                for (int i = 0; i < 10 && oscillating; i++) {
                    if (history[history.size()-1-i] != history[history.size()-1-i-period]) {
                        oscillating = false;
                    }
                }
                if (oscillating) break;  // 진동 패턴 감지 시 조기 종료
            }

            prev = alive;
            nextGenKernel<<<gridSize, blockSize, 0, str>>>(d_curr, d_next, W, H);
            hipStreamSynchronize(str);
            std::swap(d_curr, d_next);
            gen++;
        }

        int final_count = 0;
        {
            thrust::device_ptr<const int> ptr(d_curr);
            final_count = thrust::reduce(thrust::cuda::par.on(str), ptr, ptr + H * W, 0, thrust::plus<int>());
            hipStreamSynchronize(str);
        }

        d_matrix_2<int> final_board(H, W, str);
        hipMemcpyAsync(final_board.getDevPointer(), d_curr, sizeof(int) * H * W, hipMemcpyDeviceToDevice, str);
        hipStreamSynchronize(str);

    if (d_alt) hipFreeAsync(d_alt, str);
        hipStreamSynchronize(str);
        return {std::move(final_board), final_count};
    }

    void generateGameOfLifeData(int filenum, double ratio, int seed, dataset_id info) {
        int deviceCount = 0;
        hipError_t err = hipGetDeviceCount(&deviceCount);
        if (err != hipSuccess || deviceCount == 0) {
            std::cerr << "[FATAL] No CUDA device: " << hipGetErrorString(err) << std::endl;
            exit(1);
        }
        hipSetDevice(0);

        // 스트림 생성
        hipStream_t stream;
        hipStreamCreate(&stream);

        std::string datasetPath = "../" + getDatasetId(info) + "/";

        
        if (fs::exists(datasetPath)) {
            std::cout << "[INFO] Dataset directory already exists: " << datasetPath << std::endl;
            return;
        }else {
            fs::create_directories(datasetPath);
        }
        int totalFiles = filenum;
        double aliveratio = ratio;

        std::cout << "totalFiles:" << totalFiles << " (file direction: " << datasetPath << ")" << std::endl;
        std::cout << "aliveratio:" << aliveratio << std::endl;
        std::cout << "max generation:" << MAXGEN << std::endl;
        std::cout << "pattern size:" << HEIGHT << " * " << WIDTH << std::endl;
        std::cout << "board size:" << BOARDHEIGHT << " * " << BOARDWIDTH << std::endl;

        auto startTime = std::chrono::steady_clock::now();

        // 고정 시드 기반 난수 엔진 (파일 ID 오프셋으로 각 샘플을 유일화하되 결정성 유지)
        std::mt19937_64 global_gen(static_cast<uint64_t>(seed));
        std::uniform_int_distribution<int> offset_dist(0, std::numeric_limits<int>::max());

        for (int fileId = 1; fileId <= totalFiles; ++fileId) {
            // 각 샘플에 대해 고유하지만 결정적인 시드를 생성
            // 시드 충돌 줄이기 위해 64비트 혼합
            uint64_t file_seed = static_cast<uint64_t>(seed) ^ (static_cast<uint64_t>(fileId) * 0x9E3779B97F4A7C15ULL);
            // 패턴 생성에 시드 적용
            d_matrix_2<int> pattern = generateFixedRatioPatternWithSeed(
                BOARDHEIGHT, BOARDWIDTH, HEIGHT, WIDTH, aliveratio, static_cast<int>(file_seed & 0x7fffffff), stream);

            pattern.cpyToHost(stream);
            
            // 단일 시뮬레이션으로 최종 패턴과 라벨을 동시에 얻음
            int label = simulateAndLabel(pattern, fileId, stream);

            std::ofstream fout(datasetPath + "sample" + std::to_string(fileId) + ".txt");

            int startRow = (BOARDHEIGHT - HEIGHT) / 2;
            int startCol = (BOARDWIDTH - WIDTH) / 2;
            
            // GPU 작업이 완료될 때까지 대기
            hipStreamSynchronize(stream);

            // 초기 패턴 저장
            for (int i = startRow; i < startRow + HEIGHT; ++i) {
                for (int j = startCol; j < startCol + WIDTH; ++j) {
                    fout << pattern(i, j);
                }
                fout << '\n';
            }

            fout << label << '\n';
            fout << '\n';

            // 최종 패턴을 호스트로 복사 (비동기)
            //last_pattern.cpyToHost(stream);
            //hipStreamSynchronize(stream);
            //// 최종 패턴 저장
            //for(int i = 0; i < BOARDHEIGHT; i++){
            //    for(int j = 0; j < BOARDWIDTH; j++){
            //        fout << last_pattern(i, j);
            //    }
            //    fout << '\n';
            //}

            fout.close();
            
            // 진행률 표시 최적화: 매 10번째마다만 업데이트
            if (fileId % 10 == 0 || fileId == totalFiles) {
                std::string prograss_name = "GOL data generating... " + std::to_string(fileId) + "/" + std::to_string(totalFiles);
                printProgressBar(fileId, totalFiles, startTime, prograss_name);
            }
        }
        
        std::cout << std::endl << "[Done] Dataset generation complete." << std::endl;

        auto totalElapsed = std::chrono::steady_clock::now() - startTime;
        int totalSec = std::chrono::duration_cast<std::chrono::seconds>(totalElapsed).count();
        std::cout << "총 실행 시간: " << totalSec << " 초" << std::endl;

        hipStreamDestroy(stream);
    }

    std::pair<d_matrix_2<double>, d_matrix_2<double>> LoadingDataBatch(dataset_id info, hipStream_t str) {
        std::vector<std::pair<d_matrix_2<double>, d_matrix_2<double>>> temp_dataset;
        temp_dataset.reserve(5000);  // 4000개 + 여유분

        std::string datasetPath = "../" + getDatasetId(info) + "/";

        // 파일들을 읽어서 임시 벡터에 저장
        for (const auto& entry : fs::directory_iterator(datasetPath)) {
            if (entry.path().extension() != ".txt") continue;

            std::ifstream fin(entry.path());
            if (!fin) {
                std::cerr << "파일 열기 실패: " << entry.path() << '\n';
                continue;
            }

            d_matrix_2<double> input(WIDTH*HEIGHT, 1, str);
            std::string line;
            int row = 0;
            while (row < WIDTH && std::getline(fin, line)) {
                int len = std::min(HEIGHT, static_cast<int>(line.size()));
                for (int col = 0; col < len; ++col) {
                    input(row * HEIGHT + col, 0) = line[col] - '0';
                }
                row++;
            }

            int label_index = -1;
            if (std::getline(fin, line)) label_index = std::stoi(line);

            d_matrix_2<double> label(BIT_WIDTH, 1, str);
            // 1) 모두 0으로 초기화
            label.fill(0.0, str);
            // 2) 각 비트 위치에 0/1 설정 (LSB부터)
            for (int b = 0; b < BIT_WIDTH; ++b) {
                label(b, 0) = (label_index >> b) & 1;
            }

            input.cpyToDev(str);
            label.cpyToDev(str);
            temp_dataset.emplace_back(std::move(input), std::move(label));
        }

        int N = temp_dataset.size();
        std::cout << "로드된 데이터 개수: " << N << std::endl;

        // 배치 친화적인 형태로 변환: (N, features) 형태의 행렬 2개
        d_matrix_2<double> X(N, WIDTH*HEIGHT, str);  // 입력 행렬: (샘플수, 특성수)
        d_matrix_2<double> Y(N, BIT_WIDTH, str);     // 라벨 행렬: (샘플수, 클래스수)

        // 데이터 복사
        for (int i = 0; i < N; i++) {
            temp_dataset[i].first.cpyToHost(str);
            temp_dataset[i].second.cpyToHost(str);
            hipStreamSynchronize(str);

            // 입력 데이터 복사
            for (int j = 0; j < WIDTH*HEIGHT; j++) {
                X(i, j) = temp_dataset[i].first.getHostPointer()[j];
            }

            // 라벨 데이터 복사
            for (int j = 0; j < BIT_WIDTH; j++) {
                Y(i, j) = temp_dataset[i].second.getHostPointer()[j];
            }
        }

        // GPU로 전송
        X.cpyToDev(str);
        Y.cpyToDev(str);
        hipStreamSynchronize(str);

        return {std::move(X), std::move(Y)};
    }

} // namespace GOL_2
