#include "hip/hip_runtime.h"
#include <perceptron_2.hpp>
#include <utility.hpp>
#include <GOLdatabase_2.hpp>
#include <chrono>
#include <iostream>
#include <fstream>
#include <string>
#include <filesystem>

namespace fs = std::filesystem;
namespace p2 = perceptron_2;

class GOLsolver_1{
    private:
        model_id model_info;
        dataset_id using_dataset;

        p2::handleStream hs;
        p2::ActivateLayer act;
        p2::LossLayer loss;

        p2::convLayer conv1;
        p2::convLayer conv2;
        p2::convLayer conv3;

        p2::PerceptronLayer fc1;
        p2::PerceptronLayer fc2;
        p2::PerceptronLayer fc3;
        p2::PerceptronLayer fc4;
        p2::PerceptronLayer fc_out;

        int batch;
        double learning_rate;
        p2::ActType fcAct;
        p2::ActType convAct;
        p2::ActType outAct;
        p2::LossType l;
    public:
        GOLsolver_1(int bs, double lr) : batch(bs), learning_rate(lr),

         // Conv layers: 10x10 -> feature extraction
         conv1(bs, 1, 10, 10,   8, 3, 3,  0, 0,  1, 1, p2::optType::Adam, d2::InitType::He, lr, hs.model_str), // 10x10x1 -> 8x8x8
         conv2(bs, 8, 8, 8,     16, 3, 3, 0, 0,  1, 1, p2::optType::Adam, d2::InitType::He, lr, hs.model_str), // 8x8x8 -> 6x6x16
         conv3(bs, 16, 6, 6,    32, 3, 3, 0, 0,  1, 1, p2::optType::Adam, d2::InitType::He, lr, hs.model_str), // 6x6x16 -> 4x4x32

         // FC layers: flattened features -> prediction
         fc1(bs, 4*4*32, 256, p2::optType::Adam, d2::InitType::Xavier, lr, hs.model_str),     // 512 -> 256
         fc2(bs, 256, 128, p2::optType::Adam, d2::InitType::Xavier, lr, hs.model_str),         // 256 -> 128
         fc3(bs, 128, 64, p2::optType::Adam, d2::InitType::Xavier, lr, hs.model_str),          // 128 -> 64
         fc4(bs, 64, 32, p2::optType::Adam, d2::InitType::Xavier, lr, hs.model_str),          // 64 -> 32
         fc_out(bs, 32, 8, p2::optType::Adam, d2::InitType::Xavier, lr, hs.model_str)     // 32 -> 8 (output)

         {
            model_info.model_name = "mercury";
            model_info.conv_active = "LReLU";
            model_info.conv_init = "He";
            model_info.fc_active = "Tanh";
            model_info.fc_init = "Xavier";
            model_info.conv_layer_count = 3;
            model_info.fc_layer_count = 5;
            model_info.optimizer = "Adam";
            model_info.loss = "BCEWithLogits";
            model_info.epoch = 1000;
            model_info.batch_size = bs;
            model_info.learning_rate = lr;

            convAct = p2::ActType::LReLU;
            fcAct = p2::ActType::Tanh;
            l = p2::LossType::BCEWithLogits;

            if(l == p2::LossType::BCEWithLogits){
                outAct = p2::ActType::Identity;
            }else if(l == p2::LossType::CrossEntropy){
                outAct = p2::ActType::Softmax;
            }else{
                outAct = p2::ActType::LReLU;
            }

            using_dataset.seed = 54321;
            using_dataset.sample_quantity = 8000;
            using_dataset.alive_ratio = 0.3;
        }

        void genDataset(){
            GOL_2::generateGameOfLifeData(using_dataset.sample_quantity, using_dataset.alive_ratio, using_dataset.seed, using_dataset);
        }

        std::pair<d2::d_matrix_2<double>, double> forward(d2::d_matrix_2<double> X, d2::d_matrix_2<double> target, hipStream_t str = 0){
            // Conv layers with activation
            conv1.forward(X, str);
            
            conv2.forward(act.Active(conv1.getOutput(), convAct, str), str);
            
            conv3.forward(act.Active(conv2.getOutput(), convAct, str), str);
            
            // FC layers with activation
            fc1.feedforward(act.Active(conv3.getOutput(), convAct, str), str);
            
            fc2.feedforward(act.Active(fc1.getOutput(), fcAct, str), str);
            
            fc3.feedforward(act.Active(fc2.getOutput(), fcAct, str), str);

            fc4.feedforward(act.Active(fc3.getOutput(), fcAct, str), str);

            fc_out.feedforward(act.Active(fc4.getOutput(), fcAct, str), str);
            // BCEWithLogits uses raw logits (no softmax)
            auto final_output = act.Active(fc_out.getOutput(), outAct, str);

            // Loss calculation  
            double loss_val = loss.getLoss(final_output, target, l, str);
            
            return {final_output, loss_val};
        }
        
        void backward(d2::d_matrix_2<double> final_output, d2::d_matrix_2<double> target, hipStream_t str = 0){
            // Get loss gradient
            auto loss_grad = loss.getGrad(final_output, target, l, str);
            
            // Backward through FC layers (logits head uses Identity derivative)
            auto fc_out_act_deriv = act.d_Active(fc_out.getOutput(), outAct, str);
            auto delta_fc_out = fc_out.backprop(loss_grad, fc_out_act_deriv, str);

            auto fc4_act_deriv = act.d_Active(fc4.getOutput(), fcAct, str);
            auto delta_fc4 = fc4.backprop(delta_fc_out, fc4_act_deriv, str);

            auto fc3_act_deriv = act.d_Active(fc3.getOutput(), fcAct, str);
            auto delta_fc3 = fc3.backprop(delta_fc4, fc3_act_deriv, str);

            auto fc2_act_deriv = act.d_Active(fc2.getOutput(), fcAct, str);
            auto delta_fc2 = fc2.backprop(delta_fc3, fc2_act_deriv, str);
            
            auto fc1_act_deriv = act.d_Active(fc1.getOutput(), fcAct, str);
            auto delta_fc1 = fc1.backprop(delta_fc2, fc1_act_deriv, str);
            
            // Backward through conv layers
            auto conv3_act_deriv = act.d_Active(conv3.getOutput(), convAct, str);
            auto delta_conv3 = conv3.backward(delta_fc1, conv3_act_deriv, str);
            
            auto conv2_act_deriv = act.d_Active(conv2.getOutput(), convAct, str);
            auto delta_conv2 = conv2.backward(delta_conv3, conv2_act_deriv, str);
            
            auto conv1_act_deriv = act.d_Active(conv1.getOutput(), convAct, str);
            conv1.backward(delta_conv2, conv1_act_deriv, str);
        }

        bool saveModel(const std::string& filepath) const {
            std::ofstream out(filepath, std::ios::binary);
            if (!out) return false;
            const char magic[4] = {'G','O','L','1'};
            out.write(magic, sizeof(magic));
            uint32_t layerCount = 8;
            out.write(reinterpret_cast<const char*>(&layerCount), sizeof(layerCount));
            conv1.saveBinary(out);
            conv2.saveBinary(out);
            conv3.saveBinary(out);
            fc1.saveBinary(out);
            fc2.saveBinary(out);
            fc3.saveBinary(out);
            fc4.saveBinary(out);
            fc_out.saveBinary(out);
            return static_cast<bool>(out);
        }

        bool loadModel(const std::string& filepath){
            std::ifstream in(filepath, std::ios::binary);
            if (!in) return false;
            char magic[4];
            in.read(magic, sizeof(magic));
            if (!in || magic[0] != 'G' || magic[1] != 'O' || magic[2] != 'L' || magic[3] != '1') return false;
            uint32_t layerCount = 0;
            in.read(reinterpret_cast<char*>(&layerCount), sizeof(layerCount));
            if (!in || layerCount != 8) return false;
            conv1.loadBinary(in, hs.model_str);
            conv2.loadBinary(in, hs.model_str);
            conv3.loadBinary(in, hs.model_str);
            fc1.loadBinary(in, hs.model_str);
            fc2.loadBinary(in, hs.model_str);
            fc3.loadBinary(in, hs.model_str);
            fc4.loadBinary(in, hs.model_str);
            fc_out.loadBinary(in, hs.model_str);
            return static_cast<bool>(in);
        }

        void train(int epochs){
            auto start = std::chrono::steady_clock::now();
            
            // GOL 데이터 로드 (배치 형태로 직접 로드)
            auto [X, Y] = GOL_2::LoadingDataBatch(using_dataset, hs.model_str);

            int N = X.getRow();      // 전체 데이터 개수
            int input_size = X.getCol();   // 입력 크기 (100)
            int output_size = Y.getCol();  // 출력 크기 (8)
            
            std::cout << "[데이터 로드 완료] " << N << "개 샘플, 입력크기: " << input_size << ", 출력크기: " << output_size << std::endl;
            
            int B = batch;           // 배치 크기
            int num_batches = (N + B - 1) / B;  // 총 배치 수
            
            // 배치별로 데이터 미리 분할
            std::vector<d2::d_matrix_2<double>> batch_data(num_batches), batch_labels(num_batches);
            for(int i = 0; i < num_batches; ++i){
                batch_data[i] = X.getBatch(B, i*B);
                batch_labels[i] = Y.getBatch(B, i*B);
                printProgressBar(i+1, num_batches, start, "batch loading... (batch " + std::to_string(i+1) + "/" + std::to_string(num_batches) + ")");
            }
            std::cout << std::endl;
            std::cout << "[배치 로드 완료] 총 " << N << "개 데이터, " << num_batches << "개 배치" << std::endl;
            
            // Loss 데이터 저장을 위한 파일 생성
            std::string graphPath = "../graph/" + getModelId(model_info);
            fs::create_directories(graphPath);
            std::ofstream epoch_loss_file(graphPath + "/epoch_loss.txt");
            std::ofstream batch_loss_file(graphPath + "/batch_loss.txt");

            // 훈련 루프
            std::string progress_avgloss;
            for(int e = 1; e <= epochs; e++) {
                double avgloss = 0;
                
                for(int j = 0; j < num_batches; j++){
                    // 순전파
                    auto [output, loss_val] = forward(batch_data[j], batch_labels[j], hs.model_str);
                    
                    avgloss += loss_val;
                    
                    // NaN 체크
                    if(std::isnan(loss_val)){
                        std::cerr << "Loss is NaN at batch " << j+1 << ", epoch " << e << std::endl;
                        std::cerr << "Output (first 10 elements): ";
                        output.cpyToHost();
                        for(int k=0; k<std::min(10, (int)output.size()); ++k) 
                            std::cerr << output.getHostPointer()[k] << " ";
                        std::cerr << std::endl;
                        std::cerr << "Labels (first 10 elements): ";
                        batch_labels[j].cpyToHost();
                        for(int k=0; k<std::min(10, (int)batch_labels[j].size()); ++k) 
                            std::cerr << batch_labels[j].getHostPointer()[k] << " ";
                        std::cerr << std::endl;
                        throw std::runtime_error("Invalid error in loss calc.");
                    }
                    
                    // 역전파
                    backward(output, batch_labels[j], hs.model_str);
                    
                    // 배치별 loss 저장
                    batch_loss_file << e << " " << j+1 << " " << loss_val << std::endl;
                    
                    // 진행 상황 표시
                    std::string progress_batch = "batch" + std::to_string(j+1);
                    std::string progress_loss = "loss:" + std::to_string(loss_val);
                    printProgressBar(e, epochs, start, progress_avgloss + " | " + progress_batch + " 의 " + progress_loss);
                }
                
                avgloss = avgloss / static_cast<double>(num_batches);
                progress_avgloss = "[epoch" + std::to_string(e+1) + "/" + std::to_string(epochs) + "의 avgloss]:" + std::to_string(avgloss);
                
                // Epoch별 평균 loss 저장
                epoch_loss_file << e << " " << avgloss << std::endl;
                
            }
            
            // 파일 닫기
            epoch_loss_file.close();
            batch_loss_file.close();
            
            std::cout << std::endl;
            std::cout << "총 학습 시간: "
                      << std::chrono::duration_cast<std::chrono::seconds>(
                             std::chrono::steady_clock::now() - start
                         ).count() << "초" << std::endl;
        }

};


int main(){
    try {
        // CUDA 디바이스 확인
        int deviceCount = 0;
        hipError_t err = hipGetDeviceCount(&deviceCount);
        if (err != hipSuccess || deviceCount == 0) {
            std::cerr << "[FATAL] No CUDA device: " << hipGetErrorString(err) << std::endl;
            return 1;
        }
        std::cout << "CUDA devices found: " << deviceCount << std::endl;
        
        // 설정
        constexpr int BATCH_SIZE = 50;
        constexpr int EPOCHS = 1000;
        constexpr double lr = 1e-6;
        
        std::cout << "\n=== GOL CNN Solver 훈련 시작 ===" << std::endl;
        std::cout << "Batch Size: " << BATCH_SIZE << std::endl;
        std::cout << "Epochs: " << EPOCHS << std::endl;
        std::cout << "Learning Rate: " << lr << std::endl;

        // 솔버 생성
        GOLsolver_1 mercury(BATCH_SIZE, lr);

        std::cout << "\n=== 데이터셋 생성 ===" << std::endl;
        mercury.genDataset();

        std::cout << "\n=== 모델 훈련 시작 ===" << std::endl;
        // 훈련 실행
        mercury.train(EPOCHS);
        
        std::cout << "\n=== 훈련 완료! ===" << std::endl;
        
    } catch (const std::exception& e) {
        std::cerr << "Error: " << e.what() << std::endl;
        return 1;
    }
    
    return 0;
}


